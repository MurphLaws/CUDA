
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

void MatrixInit(float *M, int n, int p) {
    int i, j;
    for (i = 0; i < n; i++)
        for (j = 0; j < p; j++)
            M[i * p + j] = (static_cast<float>(rand()) / RAND_MAX) * 2 - 1;
}

void MatrixPrint(float *M, int n, int p) {
    int i, j;
    for (i = 0; i < n; i++) {
        printf("\n");
        for (j = 0; j < p; j++)
            printf("%f\t", M[i * p + j]);
    
    }
    printf("\n");
}

void MatrixAdd(float *M1, float *M2, float *Mout, int n, int p) {
    int i, j;
    for (i = 0; i < n; i++)
        for (j = 0; j < p; j++)
            Mout[i * p + j] = M1[i * p + j] + M2[i * p + j];


}


void MatrixMult(float *M1, float *M2, float *Mout, int n){
    int i, j, k;
    for (i = 0; i < n; i++)
        for (j = 0; j < n; j++) {
            Mout[i * n + j] = 0;
            for (k = 0; k < n; k++)
                Mout[i * n + j] += M1[i * n + k] * M2[k * n + j];
        }
    
}

__global__ void cudaMatrixAdd(float *M1, float *M2, float *Mout, int n, int p) {
    int i = blockIdx.x;
    int j = threadIdx.x;
    Mout[i * p + j] = M1[i * p + j] + M2[i * p + j];
}

__global__ void cudaMatrixMult(float *M1, float *M2, float *Mout, int n) {
    int i = blockIdx.x;
    int j = threadIdx.x;
    int k;
    Mout[i * n + j] = 0;
    for (k = 0; k < n; k++)
        Mout[i * n + j] += M1[i * n + k] * M2[k * n + j];
}


int CPU_test() {
    //MatrixAdd test
    int n = 3, p = 3;
    float *M1, *M2, *Mout;
    M1 = (float *)malloc(n * p * sizeof(float));
    M2 = (float *)malloc(n * p * sizeof(float));

    MatrixInit(M1, n, p);
    MatrixInit(M2, n, p);

    Mout = (float *)malloc(n * p * sizeof(float));
    MatrixAdd(M1, M2, Mout, n, p);

    //MatrixPrint(M1, n, p);
    //MatrixPrint(M2, n, p);
    //MatrixPrint(Mout, n, p);

    free(M1);
    free(M2);
    free(Mout);

    //MatrixMult test
    n = 2;
    float *M3, *M4, *Mout2;
    M3 = (float *)malloc(n * n * sizeof(float));
    M4 = (float *)malloc(n * n * sizeof(float));

    MatrixInit(M3, n, n);
    MatrixInit(M4, n, n);

    Mout2 = (float *)malloc(n * n * sizeof(float));
    MatrixMult(M3, M4, Mout2, n);

    MatrixPrint(M3, n, n);
    MatrixPrint(M4, n, n);
    MatrixPrint(Mout2, n, n);

    free(M3);
    free(M4);
    free(Mout2);

    return 0;
}

int GPUtest() {
    //cudaMatrixAdd test
    int n = 3, p = 3;
    float *M1, *M2, *Mout;
    float *d_M1, *d_M2, *d_Mout;

    M1 = (float *)malloc(n * p * sizeof(float));
    M2 = (float *)malloc(n * p * sizeof(float));

    MatrixInit(M1, n, p);
    MatrixInit(M2, n, p);

    Mout = (float *)malloc(n * p * sizeof(float));

    hipMalloc((void **)&d_M1, n * p * sizeof(float));
    hipMalloc((void **)&d_M2, n * p * sizeof(float));
    hipMalloc((void **)&d_Mout, n * p * sizeof(float));

    hipMemcpy(d_M1, M1, n * p * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_M2, M2, n * p * sizeof(float), hipMemcpyHostToDevice);

    cudaMatrixAdd<<<n, p>>>(d_M1, d_M2, d_Mout, n, p);

    hipMemcpy(Mout, d_Mout, n * p * sizeof(float), hipMemcpyDeviceToHost);

    //MatrixPrint(M1, n, p);
    //MatrixPrint(M2, n, p);

    //MatrixPrint(Mout, n, p);

    free(M1);
    free(M2);
    free(Mout);

    hipFree(d_M1);
    hipFree(d_M2);
    hipFree(d_Mout);



    // cudaMatrixMult test

    n = 2;
    float *M3, *M4, *Mout2;
    float *d_M3, *d_M4, *d_Mout2;

    M3 = (float *)malloc(n * n * sizeof(float));
    M4 = (float *)malloc(n * n * sizeof(float));

    MatrixInit(M3, n, n);
    MatrixInit(M4, n, n);

    Mout2 = (float *)malloc(n * n * sizeof(float));

    hipMalloc((void **)&d_M3, n * n * sizeof(float));
    hipMalloc((void **)&d_M4, n * n * sizeof(float));
    hipMalloc((void **)&d_Mout2, n * n * sizeof(float));

    hipMemcpy(d_M3, M3, n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_M4, M4, n * n * sizeof(float), hipMemcpyHostToDevice);

    cudaMatrixMult<<<n, n>>>(d_M3, d_M4, d_Mout2, n);

    hipMemcpy(Mout2, d_Mout2, n * n * sizeof(float), hipMemcpyDeviceToHost);

    MatrixPrint(M3, n, n);
    MatrixPrint(M4, n, n);

    MatrixPrint(Mout2, n, n);

    free(M3);
    free(M4);
    free(Mout2);

    hipFree(d_M3);
    hipFree(d_M4);
    hipFree(d_Mout2);

    return 0;

}

int main() {
    
    //CPU_test();
    GPUtest();
    return 0;
}
