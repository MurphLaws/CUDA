#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "matrix_operations.cuh"


int main(){

    float *d_raw_data, *d_C1_data, *d_S2_data, *d_C1_kernel, *d_C3_data,*d_C3_kernel, *d_S4_data, *d_C5_data, *d_C5_weights, *d_C6_data, *d_C6_weights, *d_C7_data, *d_C7_weights;



    //INPUT IMAGE
    int image_size = 10;
    float *raw_data = (float*)malloc(image_size*image_size*sizeof(float));

    MatrixInit01_2D(raw_data, image_size,image_size); // REPLACE BY REAL IMAGE

    //C1 DECLARATION

    int C1_filters = 4;
    int output_size_C1 = 8;

    float *C1_data  = (float*)malloc(C1_filters*output_size_C1*output_size_C1*sizeof(float));
    MatrixInit0_3D(C1_data , C1_filters, output_size_C1, output_size_C1);

    int kernel_size = 3;
    float *C1_kernel = (float*)malloc(C1_filters*kernel_size*kernel_size*sizeof(float));
    MatrixInitFilter(C1_kernel, C1_filters, kernel_size, kernel_size); //CHANGE FOR FILE

    //S2 DECLARATION

    int output_size_S2 = output_size_C1/2;
    float *S2_data = (float*)malloc(C1_filters*output_size_S2*output_size_S2*sizeof(float));
    MatrixInit0_3D(S2_data, C1_filters, output_size_S2, output_size_S2);

    //C3 DECLARATION

    int C3_filters = 3*C1_filters;
    int output_size_C3 = 2;
    float *C3_data = (float*)malloc(C3_filters*output_size_C3*output_size_C3*sizeof(float));
    MatrixInit0_3D(C3_data, C3_filters, output_size_C3, output_size_C3); 

    float *C3_kernel = (float*)malloc(C3_filters*C1_filters*kernel_size*kernel_size*sizeof(float));
    MatrixInitFilter(C3_kernel, C3_filters, kernel_size, kernel_size); //CHANGE FOR FILE


    //S4 DECLARATION

    int output_size_S4 = output_size_C3/2;
    float *S4_data = (float*)malloc(C3_filters*output_size_S4*output_size_S4*sizeof(float));
    MatrixInit0_3D(S4_data, C3_filters, output_size_S4, output_size_S4);

    // C5 DECLARATION [Dense Layer]

    int C5_input = 12;
    int C5_output = 5;

    float *C5_data = (float*)malloc(C5_output*sizeof(float));
    MatrixInit0(C5_data, C5_output);

    float *C5_weights = (float*)malloc(C5_input*C5_output*sizeof(float));
    MatrixInitFilter(C5_weights, C5_output, C5_input, 1); //CHANGE FOR FILE

    // C6

    int C6_input = 5;
    int C6_output = 3;

    float *C6_data = (float*)malloc(C6_output*sizeof(float));
    MatrixInit0(C6_data, C6_output);

    float *C6_weights = (float*)malloc(C6_input*C6_output*sizeof(float));
    MatrixInitFilter(C6_weights, C6_output, C6_input, 1); //CHANGE FOR FILE

    // C7

    int C7_input = 3;
    int C7_output = 1;

    float *C7_data = (float*)malloc(C7_output*sizeof(float));
    MatrixInit0(C7_data, C7_output);

    float *C7_weights = (float*)malloc(C7_input*C7_output*sizeof(float));
    MatrixInitFilter(C7_weights, C7_output, C7_input, 1); //CHANGE FOR FILE






    

    hipMalloc((void**)&d_raw_data, image_size*image_size*sizeof(float));
    hipMalloc((void**)&d_C1_data, C1_filters*output_size_C1*output_size_C1*sizeof(float));
    hipMalloc((void**)&d_C1_kernel, C1_filters*kernel_size*kernel_size*sizeof(float));
    hipMalloc((void**)&d_S2_data, C1_filters*output_size_S2*output_size_S2*sizeof(float));
    hipMalloc((void**)&d_C3_kernel, C3_filters*C1_filters*kernel_size*kernel_size*sizeof(float));
    hipMalloc((void**)&d_C3_data, C3_filters*output_size_C3*output_size_C3*sizeof(float));
    hipMalloc((void**)&d_S4_data, C3_filters*output_size_S4*output_size_S4*sizeof(float));
    hipMalloc((void**)&d_C5_data, C5_output*sizeof(float));
    hipMalloc((void**)&d_C5_weights, C5_input*C5_output*sizeof(float));
    hipMalloc((void**)&d_C6_data, C6_output*sizeof(float));
    hipMalloc((void**)&d_C6_weights, C6_input*C6_output*sizeof(float));
    hipMalloc((void**)&d_C7_data, C7_output*sizeof(float));
    hipMalloc((void**)&d_C7_weights, C7_input*C7_output*sizeof(float));

    hipMemcpy(d_raw_data, raw_data, image_size*image_size*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_C1_data, C1_data, C1_filters*output_size_C1*output_size_C1*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_C1_kernel, C1_kernel, C1_filters*kernel_size*kernel_size*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_S2_data, S2_data, C1_filters*output_size_S2*output_size_S2*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_C3_data, C3_data, C3_filters*output_size_C3*output_size_C3*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_C3_kernel, C3_kernel, C3_filters*C1_filters*kernel_size*kernel_size*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_S4_data, S4_data, C3_filters*output_size_S4*output_size_S4*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_C5_data, C5_data, C5_output*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_C5_weights, C5_weights, C5_input*C5_output*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_C6_data, C6_data, C6_output*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_C6_weights, C6_weights, C6_input*C6_output*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_C7_data, C7_data, C7_output*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_C7_weights, C7_weights, C7_input*C7_output*sizeof(float), hipMemcpyHostToDevice);


    //C1
    convolution<<<1,C1_filters>>>(d_raw_data, image_size, kernel_size, C1_filters, d_C1_kernel, d_C1_data);
    //C2
    dim3 grid(C1_filters);
    dim3 block(output_size_S2, output_size_S2);
    avgPooling<<<grid,block>>>(d_C1_data, d_S2_data, C1_filters, output_size_C1, output_size_C1, 2);
    //C3
    convolution<<<1,C3_filters*2>>>(d_S2_data, output_size_S2, kernel_size, C3_filters, d_C3_kernel, d_C3_data);

    //S4
    dim3 grid2(C3_filters);
    dim3 block2(output_size_S4, output_size_S4);
    avgPooling<<<grid2,block2>>>(d_C3_data, d_S4_data, C3_filters, output_size_C3, output_size_C3, 2);

    //C5
    denseLayer<<<1,C5_output>>>(d_S4_data, d_C5_weights, d_C5_data, C5_input, C5_output);

    //C6
    denseLayer<<<1,C6_output>>>(d_C5_data, d_C6_weights, d_C6_data, C6_input, C6_output);

    //C7
    denseLayer<<<1,C7_output>>>(d_C6_data, d_C7_weights, d_C7_data, C7_input, C7_output);



    hipMemcpy(C1_data, d_C1_data, C1_filters*output_size_C1*output_size_C1*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(S2_data, d_S2_data, C1_filters*output_size_S2*output_size_S2*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(C3_data, d_C3_data, C3_filters*output_size_C3*output_size_C3*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(S4_data, d_S4_data, C3_filters*output_size_S4*output_size_S4*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(C5_data, d_C5_data, C5_output*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(C6_data, d_C6_data, C6_output*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(C7_data, d_C7_data, C7_output*sizeof(float), hipMemcpyDeviceToHost);


    
    MatrixPrint3D(S2_data, C1_filters, output_size_S2, output_size_S2);
    printf("-----------------------------------------\n");
    MatrixPrint3D(C3_data, C3_filters, output_size_C3, output_size_C3);
    printf("-----------------------------------------\n");
    MatrixPrint3D(S4_data, C3_filters, output_size_S4, output_size_S4);
    printf("-----------------------------------------\n");
    MatrixPrint(C5_data, C5_output, 1);
    printf("-----------------------------------------\n");
    MatrixPrint(C6_data, C6_output, 1);
    printf("-----------------------------------------\n");
    MatrixPrint(C7_data, C7_output, 1);
    printf("-----------------------------------------\n");
    


}