#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "matrix_operations.cuh"


#define WIDTH 28
#define HEIGHT 28


void MatrixInit(float *M, int n, int p) {
    int i, j;
    for (i = 0; i < n; i++)
        for (j = 0; j < p; j++)
            M[i * p + j] = (static_cast<float>(rand()) / RAND_MAX) * 2 - 1;
}



void MatrixInit01_2D(float *M, int n, int p) {
    int i, j;
    for (i = 0; i < n; i++)
        for (j = 0; j < p; j++)
            M[i * p + j] = (static_cast<float>(rand()) / RAND_MAX);
}
void MatrixInit0_3D(float *M, int n, int p, int q) {
    int i, j, k;
    for (i = 0; i < n; i++)
        for (j = 0; j < p; j++)
            for (k = 0; k < q; k++)
                M[i * p * q + j * q + k] = 0;
}



void MatrixInit01_3D(float *M, int n, int p, int q) {
    int i, j, k;
    for (i = 0; i < n; i++)
        for (j = 0; j < p; j++)
            for (k = 0; k < q; k++)
                M[i * p * q + j * q + k] = (static_cast<float>(rand()) / RAND_MAX);
}

void MatrixInit0(float *M, int n) {
    int i;
    for (i = 0; i < n; i++)
        M[i] = 0;
}


void MatrixInitFilter(float *M, int n, int p, int q) {
    int i, j, k;
    for (i = 0; i < n; i++)
        for (j = 0; j < p; j++)
            for (k = 0; k < q; k++)
                M[i * p * q + j * q + k] = (static_cast<float>(rand()) / RAND_MAX) * 2 - 1;
}

void MatrixPrint(float *M, int n, int p) {
    int i, j;
    for (i = 0; i < n; i++) {
        printf("\n");
        for (j = 0; j < p; j++)
            printf("%f\t", M[i * p + j]);
    
    }
    printf("\n");
}

void MatrixAdd(float *M1, float *M2, float *Mout, int n, int p) {
    int i, j;
    for (i = 0; i < n; i++)
        for (j = 0; j < p; j++)
            Mout[i * p + j] = M1[i * p + j] + M2[i * p + j];


}


void MatrixMult(float *M1, float *M2, float *Mout, int n){
    int i, j, k;
    for (i = 0; i < n; i++)
        for (j = 0; j < n; j++) {
            Mout[i * n + j] = 0;
            for (k = 0; k < n; k++)
                Mout[i * n + j] += M1[i * n + k] * M2[k * n + j];
        }
    
}




void MatrixPrint3D(float *M, int n, int p, int q) {
    int i, j, k;
    for (i = 0; i < n; i++) {
        printf("\n");
        for (j = 0; j < p; j++) {
            printf("\n");
            for (k = 0; k < q; k++)
                printf("%f\t", M[i * p * q + j * q + k]);
        }
        printf("\n");
    }
}

__global__ void cudaMatrixAdd(float *M1, float *M2, float *Mout, int n, int p) {
    int i = blockIdx.x;
    int j = threadIdx.x;
    Mout[i * p + j] = M1[i * p + j] + M2[i * p + j];
}

__global__ void cudaMatrixMult(float *M1, float *M2, float *Mout, int n) {
    int i = blockIdx.x;
    int j = threadIdx.x;
    int k;
    Mout[i * n + j] = 0;
    for (k = 0; k < n; k++)
        Mout[i * n + j] += M1[i * n + k] * M2[k * n + j];
}


__global__ void cudaMatrixSum(float *M, float *sum, int n, int p) {
    int i = blockIdx.x;
    int j = threadIdx.x;
    sum[i * p + j] = 0;
    for (int k = 0; k < n; k++)
        sum[i * p + j] += M[i * p + k];
}




__global__ void convolution(float *input, int input_size, int kernel_size, int number_of_filters, float *kernel, float *output) {
    int output_size = input_size - kernel_size + 1;
    int i = threadIdx.x;
    int j = threadIdx.y; 
 

    for (int f = 0; f < number_of_filters; f++) {
      
        for (int k = 0; k < output_size; k++) {
            for (int l = 0; l < output_size; l++) {
                float sum = 0;

                
                for (int m = 0; m < kernel_size; m++) {
                    for (int n = 0; n < kernel_size; n++) {
                        int input_offset = (i + k + m) * input_size + (j + l + n);
                        int kernel_offset = f * kernel_size * kernel_size + m * kernel_size + n;
                        sum += input[input_offset] * kernel[kernel_offset];
                    }
                }

                int output_offset = f * output_size * output_size + k * output_size + l;
                output[output_offset] = tanhf(sum);
            }
        }
    }}


__global__ void avgPooling(float *M, float *P, int n, int p, int q, int poolSize) {
    int i = blockIdx.x;
    int j = threadIdx.y * poolSize;
    int k = threadIdx.x * poolSize;

    if (i < n && j < p && k < q) {
        float sum = 0.0;
        for (int m = 0; m < poolSize && j + m < p; m++) {
            for (int l = 0; l < poolSize && k + l < q; l++) {
                sum += M[i * p * q + (j + m) * q + (k + l)];
            }
        }
        P[i * (p/poolSize) * (q/poolSize) + (j/poolSize) * (q/poolSize) + (k/poolSize)] = sum / (poolSize * poolSize);
    }
}
 



__device__ float softmax(float x, float *arr, int n) {
    float sum = 0.0;
    for (int i = 0; i < n; i++)
        sum += expf(arr[i]);
    return expf(x) / sum;
}

//For the dense layers, considering we have three of those, with two different
// activation functions, we have to use a conditional  to choose the activation function

__global__ void denseLayer(float *input, float *weights, float *output, int n, int p, Activation activation) {
    int i = blockIdx.x;
    int j = threadIdx.x;
    output[i * p + j] = 0;
    for (int k = 0; k < n; k++)
        output[i * p + j] += input[i * n + k] * weights[k * p + j];

   
    if (activation == TANH) {
        output[i * p + j] = tanhf(output[i * p + j]);
    } else if (activation == SOFTMAX) {
        output[i * p + j] = softmax(output[i * p + j], output + i * p, p);
    }
}

//this function is inspired in the provided MNIST pinrint function
float *generateGrayscaleImage(int imageIndex) {
    int i, j;
    float *img;
    unsigned int magic, nbImg, nbRows, nbCols;
    unsigned char val;
    FILE *fptr;

 
    img = (float *)malloc(HEIGHT * WIDTH * sizeof(float));
 
    if ((fptr = fopen("train-images.idx3-ubyte", "rb")) == NULL) {
        printf("Can't open file");
        exit(1);
    }
 
    fread(&magic, sizeof(int), 1, fptr);
    fread(&nbImg, sizeof(int), 1, fptr);
    fread(&nbRows, sizeof(int), 1, fptr);
    fread(&nbCols, sizeof(int), 1, fptr);
 
    if (imageIndex < 0 || imageIndex >= nbImg) {
        printf("Invalid image index");
        exit(1);
    }

 
    fseek(fptr, 16 + imageIndex * HEIGHT * WIDTH * sizeof(unsigned char), SEEK_SET);
 
    for (i = 0; i < HEIGHT; i++) {
        for (j = 0; j < WIDTH; j++) {
            fread(&val, sizeof(unsigned char), 1, fptr);
            img[i * WIDTH + j] = (float)val;
        }
    }
 
    fclose(fptr);

    return img;
}

void charBckgrndPrint(const char *str, int rgb[3])
{
    printf("\033[48;2;%d;%d;%dm", rgb[0], rgb[1], rgb[2]);
    printf("%s\033[0m", str);
}

void printGrayscaleImage(int height, int width, float *img) {
    int row, col;
    const char *str = "  ";
    for (row = 0; row < height; row++) {
        for (col = 0; col < width; col++) {
            float pixel_value = img[row * width + col];
            int rounded_pixel = (int)pixel_value;
            int grayscale_rgb[3] = {rounded_pixel, rounded_pixel, rounded_pixel};
            charBckgrndPrint(str, grayscale_rgb);
        }
        printf("\n");
    }
}


//This function heps us to read the exported .txt weights from the python script
void readArrayFromFile(const char* filename, float* array, int size) {
    FILE* file = fopen(filename, "r");
    if (file == NULL) {
        fprintf(stderr, "Error opening file: %s\n", filename);
        exit(EXIT_FAILURE);
    }

    for (int i = 0; i < size; ++i) {
        if (fscanf(file, "%f", &array[i]) != 1) {
            fprintf(stderr, "Error reading from file: %s\n", filename);
            fclose(file);
            exit(EXIT_FAILURE);
        }
    }

    fclose(file);
}