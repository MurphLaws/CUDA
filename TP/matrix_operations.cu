#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "matrix_operations.cuh"

/*
L'architecture du réseau LeNet-5 est composé de plusieurs couches :

Layer 1- Couche d'entrée de taille 32x32 correspondant à la taille des images de la base de donnée MNIST

Layer 2- Convolution avec 6 noyaux de convolution de taille 5x5. La taille résultantes est donc de 6x28x28.

Layer 3- Sous-échantillonnage d'un facteur 2. La taille résultantes des données est donc de 6x14x14.

*/

void MatrixInit(float *M, int n, int p) {
    int i, j;
    for (i = 0; i < n; i++)
        for (j = 0; j < p; j++)
            M[i * p + j] = (static_cast<float>(rand()) / RAND_MAX) * 2 - 1;
}



void MatrixInit01_2D(float *M, int n, int p) {
    int i, j;
    for (i = 0; i < n; i++)
        for (j = 0; j < p; j++)
            M[i * p + j] = (static_cast<float>(rand()) / RAND_MAX);
}

//Declare the method MatrixInitFilter that initialize a 3d matric of size n*p*q with 0 values

void MatrixInit0_3D(float *M, int n, int p, int q) {
    int i, j, k;
    for (i = 0; i < n; i++)
        for (j = 0; j < p; j++)
            for (k = 0; k < q; k++)
                M[i * p * q + j * q + k] = 0;
}


//Declare the method MatrixInit01_3D that initialize a 3d matric of size n*p*q with random valñues between 0 and 1

void MatrixInit01_3D(float *M, int n, int p, int q) {
    int i, j, k;
    for (i = 0; i < n; i++)
        for (j = 0; j < p; j++)
            for (k = 0; k < q; k++)
                M[i * p * q + j * q + k] = (static_cast<float>(rand()) / RAND_MAX);
}

//Define a method for 1D arrays initialization with 0 values

void MatrixInit0(float *M, int n) {
    int i;
    for (i = 0; i < n; i++)
        M[i] = 0;
}

//Declare the method MatrixInitFilter that initialize a 3d matric of size n*p*q with random valñues between -1 and 1

void MatrixInitFilter(float *M, int n, int p, int q) {
    int i, j, k;
    for (i = 0; i < n; i++)
        for (j = 0; j < p; j++)
            for (k = 0; k < q; k++)
                M[i * p * q + j * q + k] = (static_cast<float>(rand()) / RAND_MAX) * 2 - 1;
}

void MatrixPrint(float *M, int n, int p) {
    int i, j;
    for (i = 0; i < n; i++) {
        printf("\n");
        for (j = 0; j < p; j++)
            printf("%f\t", M[i * p + j]);
    
    }
    printf("\n");
}

void MatrixAdd(float *M1, float *M2, float *Mout, int n, int p) {
    int i, j;
    for (i = 0; i < n; i++)
        for (j = 0; j < p; j++)
            Mout[i * p + j] = M1[i * p + j] + M2[i * p + j];


}


void MatrixMult(float *M1, float *M2, float *Mout, int n){
    int i, j, k;
    for (i = 0; i < n; i++)
        for (j = 0; j < n; j++) {
            Mout[i * n + j] = 0;
            for (k = 0; k < n; k++)
                Mout[i * n + j] += M1[i * n + k] * M2[k * n + j];
        }
    
}


//Define a print 3d matrix method that prints a 3d matrix of size n*p*q, printing each matrix of size p*q separated 
// by a line


void MatrixPrint3D(float *M, int n, int p, int q) {
    int i, j, k;
    for (i = 0; i < n; i++) {
        printf("\n");
        for (j = 0; j < p; j++) {
            printf("\n");
            for (k = 0; k < q; k++)
                printf("%f\t", M[i * p * q + j * q + k]);
        }
        printf("\n");
    }
}

__global__ void cudaMatrixAdd(float *M1, float *M2, float *Mout, int n, int p) {
    int i = blockIdx.x;
    int j = threadIdx.x;
    Mout[i * p + j] = M1[i * p + j] + M2[i * p + j];
}

__global__ void cudaMatrixMult(float *M1, float *M2, float *Mout, int n) {
    int i = blockIdx.x;
    int j = threadIdx.x;
    int k;
    Mout[i * n + j] = 0;
    for (k = 0; k < n; k++)
        Mout[i * n + j] += M1[i * n + k] * M2[k * n + j];
}

//Define a method that sum all the elements into a matrix

__global__ void cudaMatrixSum(float *M, float *sum, int n, int p) {
    int i = blockIdx.x;
    int j = threadIdx.x;
    sum[i * p + j] = 0;
    for (int k = 0; k < n; k++)
        sum[i * p + j] += M[i * p + k];
}




__global__ void convolution(float *input, int input_size, int kernel_size, int number_of_filters, float *kernel, float *output) {
    int output_size = input_size - kernel_size + 1;
    int i = threadIdx.x; // Current row index
    int j = threadIdx.y; // Current column index
 

    // Iterate over filters
    for (int f = 0; f < number_of_filters; f++) {
        // Iterate over output positions
        for (int k = 0; k < output_size; k++) {
            for (int l = 0; l < output_size; l++) {
                float sum = 0;

                // Perform convolution operation
                for (int m = 0; m < kernel_size; m++) {
                    for (int n = 0; n < kernel_size; n++) {
                        int input_offset = (i + k + m) * input_size + (j + l + n);
                        int kernel_offset = f * kernel_size * kernel_size + m * kernel_size + n;
                        sum += input[input_offset] * kernel[kernel_offset];
                    }
                }

                int output_offset = f * output_size * output_size + k * output_size + l;
                output[output_offset] = tanhf(sum);
            }
        }
    }}


__global__ void avgPooling(float *M, float *P, int n, int p, int q, int poolSize) {
    int i = blockIdx.x;
    int j = threadIdx.y * poolSize;
    int k = threadIdx.x * poolSize;

    if (i < n && j < p && k < q) {
        float sum = 0.0;
        for (int m = 0; m < poolSize && j + m < p; m++) {
            for (int l = 0; l < poolSize && k + l < q; l++) {
                sum += M[i * p * q + (j + m) * q + (k + l)];
            }
        }
        P[i * (p/poolSize) * (q/poolSize) + (j/poolSize) * (q/poolSize) + (k/poolSize)] = sum / (poolSize * poolSize);
    }
}

__global__ void convolution3D(float *input, int input_filters, int input_size, int kernel_size, int number_of_filters, float *kernel, float *output) {
    int output_size = input_size - kernel_size + 1;
    int f_out = threadIdx.x; // Current filter index of output
    int f_in = threadIdx.y; // Current filter index of input

    // Iterate over output positions
    for (int m = 0; m < output_size; m++) {
        for (int n = 0; n < output_size; n++) {
            float sum = 0;

            // Perform convolution operation for each filter
            for (int x = 0; x < kernel_size; x++) {
                for (int y = 0; y < kernel_size; y++) {
                    int input_offset = (f_in * input_size + m + x) * input_size + n + y;
                    int kernel_offset = (f_out * input_filters * kernel_size * kernel_size) + (f_in * kernel_size * kernel_size) + (x * kernel_size) + y;
                    sum += input[input_offset] * kernel[kernel_offset];
                }
            }
            int output_offset = (f_out * output_size + m) * output_size + n;
            output[output_offset] = sum;
        }
    }
}

//Define a denseLayer method, that takes a 1D array of size n as input, a 2D array of size n*p as weights and a 1D array of size p as output. Also takes n and p as parameters

__global__ void denseLayer(float *input, float *weights, float *output, int n, int p) {
    int i = blockIdx.x;
    int j = threadIdx.x;
    output[i * p + j] = 0;
    for (int k = 0; k < n; k++)
        output[i * p + j] += input[i * n + k] * weights[k * p + j];
}